//  nvcc -arch=sm_35 -rdc=true cuda-strip-cluster.cu -o cuda_strip 


#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <mm_malloc.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#define IDEAL_ALIGNMENT 64
using detId_t = uint32_t;
using fedId_t = uint16_t;
using fedCh_t = uint8_t;

#define ChannelThreshold 2.0
#define SeedThreshold 3.0
#define ClusterThresholdSquared 25.0
#define MaxSequentialHoles 0
#define MaxSequentialBad 1
#define MaxAdjacentBad 0
#define minGoodCharge 1620.0
#define RemoveApvShots true
//float ChannelThreshold = 2.0, SeedThreshold = 3.0, ClusterThresholdSquared = 25.0;
//uint8_t MaxSequentialHoles = 0, MaxSequentialBad = 1, MaxAdjacentBad = 0;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void clusterChecker(int nSeedStripsNC,int* clusterLastIndexLeft, int* clusterLastIndexRight, uint16_t* adc, float* clusterNoiseSquared,float* gain, uint8_t* clusterADCs, bool* trueCluster)
{
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
  while(i<nSeedStripsNC){
//	printf("cluster: %d\n",i);
  //for (int i=0; i<nSeedStripsNC; i++){
    int left=clusterLastIndexLeft[i];
    int right=clusterLastIndexRight[i];
    int size=right-left+1;
    int adcsum = 0;
    for (int j=0; j<size; j++) {
      adcsum += (int)adc[left+j];
    }
    bool noiseSquaredPass = clusterNoiseSquared[i]*ClusterThresholdSquared <= ((float)(adcsum)*float(adcsum));
    bool chargePerCMPass = (float)(adcsum)/0.047f > minGoodCharge;
    if (noiseSquaredPass&&chargePerCMPass) {
      for (int j=0; j<size; j++){
      uint8_t adc_j = adc[left+j];
      float gain_j = gain[left+j];
      auto charge = int( float(adc_j)/gain_j + 0.5f );
      if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
      clusterADCs[j*nSeedStripsNC+i] = adc_j;
      }
      trueCluster[i] = true;
    }
  //}
   i += blockDim.x*gridDim.x;
}
}


__global__ void findBoundries(int nStrips, int nSeedStripsNC,int* seedStripsNCIndex, float* clusterNoiseSquared,uint16_t* stripId,int* clusterLastIndexLeft,int* clusterLastIndexRight,uint16_t* adc,float* noise)
{
//printf("test1: %d\n",nSeedStripsNC);
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
//printf("test12: %d\n",i);
  while(i<nSeedStripsNC){
  //printf("test1: %d\n",i);
    clusterNoiseSquared[i] = 0.0;
//printf("test2:\n");
    int index=seedStripsNCIndex[i];
//printf("test3:\n");
    clusterLastIndexLeft[i] = index;
    clusterLastIndexRight[i] = index;
//printf("test4:\n");
    //uint8_t adc_i = adc[index];
    float noise_i = noise[index];
    //printf("test1: %d \n",i);
    clusterNoiseSquared[i] += noise_i*noise_i;
    //printf("noise i n: %d %f\n",i, noise_i*noise_i);
    // find left boundary
    int testIndexL=index-1;
    //printf("test Index: %d %d \n",i, testIndex);
    while(index>0&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndexL]-1)>=0)&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndexL]-1)<=MaxSequentialHoles)){

    //printf("test2:\n");
      float testNoise = noise[testIndexL];
    //printf("testnoise: %d\n",testNoise);
      uint8_t testADC = adc[testIndexL];
    //printf("testAdc: %d\n",testADC);
      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
        --clusterLastIndexLeft[i];
        clusterNoiseSquared[i] += testNoise*testNoise;
      }
      --testIndexL;
    }

    // find right boundary
    int testIndexR=index+1;
   // printf("test Index: %d %f \n",i, testIndexR);
    while(testIndexR<nStrips&&((stripId[testIndexR]-stripId[clusterLastIndexRight[i]]-1)>=0)&&((stripId[testIndexR]-stripId[clusterLastIndexRight[i]]-1)<=MaxSequentialHoles)){

   // printf("test3\n");
      float testNoise = noise[testIndexR];
   // printf("testnoise: %f\n",testNoise);
      uint8_t testADC = adc[testIndexR];
   // printf("testAdc: %f\n",testADC);
      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
        ++clusterLastIndexRight[i];
        clusterNoiseSquared[i] += testNoise*testNoise;
      }
      ++testIndexR;
    }
   i += blockDim.x*gridDim.x;
  }
}

/*
__global__ void getNCSeedStrips(const int nStrips,const float* noise_d,const uint16_t* adc,int* nSeedStripsNC_old)
{

  // find the seed strips
   unsigned int in = threadIdx.x + blockIdx.x*blockDim.x+1;
while(in<nStrips){
    float noise_i = noise_d[in];
    uint8_t adc_i = static_cast<uint8_t>(adc[in]);
    float noise_iR = noise_d[in-1];
    uint8_t adc_iR = static_cast<uint8_t>(adc[in-1]);
    nSeedStripsNC_old[in] = ((adc_iR >= static_cast<uint8_t>( noise_iR * SeedThreshold)) && (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)));
   // printf("xxx: %d %d\n",in, ((adc_iR >= static_cast<uint8_t>( noise_iR * SeedThreshold)) && (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold))));

    in = in + blockDim.x*gridDim.x;
  }
}

*/
__global__ void getNCSeedStrips(const int nStrips,const float* noise,const uint16_t* adc,const uint16_t* stripId, int* seedStripMask, int* seedStripNCMask)
{
   unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
  
while(i<nStrips){
	//printf("y: %d\n",i);
    float noise_i = noise[i];
    uint8_t adc_i = static_cast<uint8_t>(adc[i]);
    seedStripMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? true:false;
 //   printf("test i n a b: %d %f %d %d\n",i,noise_i,adc_i,seedStripMask[i]); 
  //  nSeedStrips += static_cast<int>(seedStripMask[i]);
  //  in += blockDim.x*gridDim.x;

//__syncthreads();
  //int nSeedStripsNC=0;
   //unsigned int in = threadIdx.x + blockIdx.x*blockDim.x+1;
  //seedStripNCMask[0] = seedStripMask[0];
  //if (seedStripNCMask[0]) nSeedStripsNC++;
//#pragma omp parallel for reduction(+:nSeedStripsNC)

 i += blockDim.x*gridDim.x;
}
}
__global__ void getNCSeedStrips1(const int nStrips,const uint16_t* stripId, int* seedStripMask, int* seedStripNCMask)
{
 unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
 if(i== 0){
 seedStripNCMask[0] = seedStripMask[0];
}
else{ 
while(i<nStrips){
//	printf("%d\n",i);
        seedStripNCMask[i] == false;
    if (seedStripMask[i] == true) {
      if (stripId[i]-stripId[i-1]!=1||((stripId[i]-stripId[i-1]==1)&&!seedStripMask[i-1])) {
        seedStripNCMask[i] = true;
       // nSeedStripsNC += static_cast<int>(seedStripNCMask[i]);
      }
    }

 i += blockDim.x*gridDim.x;
}
}
}
/*
if(i!=0){
    seedStripNCMask[i] = false;
    if (seedStripMask[i] == true) { 
      if (stripId[i]-stripId[i-1]!=1||((stripId[i]-stripId[i-1]==1)&&!(adc[i] >= static_cast<uint8_t>( noise[i-1] * SeedThreshold)) )) {
        seedStripNCMask[i] = true;
    //    nSeedStripsNC += static_cast<int>(seedStripNCMask[i]);
      }
    } 
  }
else{seedStripNCMask[0] = seedStripMask[0];}
//printf("NCMask i n: %d %d\n",i, seedStripNCMask[i]);
*/

int main()
{

  double start, end;
  struct timeval timecheck;

  int max_strips = 1400000;
  detId_t *detId = (detId_t *)_mm_malloc(max_strips*sizeof(detId_t), IDEAL_ALIGNMENT);
  fedId_t *fedId = (fedId_t *)_mm_malloc(max_strips*sizeof(fedId_t), IDEAL_ALIGNMENT);
  fedCh_t *fedCh = (fedCh_t *)_mm_malloc(max_strips*sizeof(fedCh_t), IDEAL_ALIGNMENT);
  uint16_t *stripId = (uint16_t *)_mm_malloc(max_strips*sizeof(uint16_t), IDEAL_ALIGNMENT);
  uint16_t *adc = (uint16_t *)_mm_malloc(max_strips*sizeof(uint16_t), IDEAL_ALIGNMENT);
  float *noise = (float *)_mm_malloc(max_strips*sizeof(float), IDEAL_ALIGNMENT);
  float *gain = (float *)_mm_malloc(max_strips*sizeof(float), IDEAL_ALIGNMENT);
  bool *bad = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);
  //bool *seedStripMask = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);
  //bool *seedStripNCMask = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);

  //gpuErrchk((cudaMallocManaged((void**)adc,max_strips*sizeof(uint16_t))));
  //gpuErrchk((cudaMallocManaged((void**)noise,max_strips*sizeof(float))));

  // read in the data
  std::ifstream digidata_in("digidata.bin", std::ofstream::in | std::ios::binary);
  int i=0;
  while (digidata_in.read((char*)&detId[i], sizeof(detId_t)).gcount() == sizeof(detId_t)) {
    //digidata_in.read((char*)&fedId[i], sizeof(fedId_t));
    //digidata_in.read((char*)&fedCh[i], sizeof(fedCh_t));
    digidata_in.read((char*)&stripId[i], sizeof(uint16_t));
    digidata_in.read((char*)&adc[i], sizeof(uint16_t));
    digidata_in.read((char*)&noise[i], sizeof(float));
    digidata_in.read((char*)&gain[i], sizeof(float));
    digidata_in.read((char*)&bad[i], sizeof(bool));
    if (bad[i])
      std::cout<<"index "<<i<<" detid "<<detId[i]<<" stripId "<<stripId[i]<<
        " adc "<<adc[i]<<" noise "<<noise[i]<<" gain "<<gain[i]<<" bad "<<bad[i]<<std::endl;
    i++;
  }
  int nStrips=i;
  
  gettimeofday(&timecheck, NULL);
  start = (double)timecheck.tv_sec *1000 + (double)timecheck.tv_usec /1000;

//  float ChannelThreshold = 2.0, SeedThreshold = 3.0, ClusterThresholdSquared = 25.0;
//  uint8_t MaxSequentialHoles = 0, MaxSequentialBad = 1, MaxAdjacentBad = 0;
//  bool RemoveApvShots = true;
//  float minGoodCharge = 1620.0;
  int nSeedStripsNC=0;
  int* nSeedStripsNC_d;
  int* nSeedStripsNC_out;
  //cudaMallocManaged((void*)nSeedStripsNC,sizeof(int));

//
//  // find the seed strips
//  int nSeedStrips=0;
//  for (int i=0; i<nStrips; i++) {
//    float noise_i = noise[i];
//    uint8_t adc_i = static_cast<uint8_t>(adc[i]);
//    seedStripMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? true:false;
//    if (seedStripMask[i] == true) nSeedStrips++;
//  }
//
//  int nSeedStripsNC=0;
//  for (int i=0; i<nStrips; i++) {
//    if (seedStripMask[i] == true) {
//      if (stripId[i]-stripId[i-1]!=1) {
//	seedStripNCMask[i] = true;
//	nSeedStripsNC++;
//      }
//    }
//  }
  float* noise_d;
  float* gain_d;
  uint16_t* adc_d;
  uint16_t* stripId_d;
  hipMalloc((void**)&noise_d, max_strips*sizeof(float));
  hipMalloc((void**)&gain_d, max_strips*sizeof(float));
  hipMalloc((void**)&adc_d, max_strips*sizeof(uint16_t));
  hipMalloc((void**)&stripId_d, max_strips*sizeof(uint16_t));
  //cudaMalloc((void**)&nSeedStripsNC_d, max_strips*sizeof(int));
  hipMallocManaged((void**)&nSeedStripsNC_d, nStrips*sizeof(int));
  //cudaalloc((void**)&nSeedStripsNC_out, max_strips*sizeof(int));
  //cudaMemcpy(&nSeedStripsNC_d, &nSeedStripsNC,sizeof(nSeedStripsNC),cudaMemcpyHostToDevice);
  hipMemcpy(noise_d, noise,max_strips*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(gain_d, gain,max_strips*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(adc_d, adc,max_strips*sizeof(uint16_t),hipMemcpyHostToDevice);
  hipMemcpy(stripId_d, stripId,max_strips*sizeof(uint16_t),hipMemcpyHostToDevice);

  int* seedStripMask;
  int* seedStripsNCMask;
  hipMallocManaged((void**)&seedStripMask, nStrips*sizeof(int));
  hipMallocManaged((void**)&seedStripsNCMask, nStrips*sizeof(int));

//getNCSeedStrips<<<32,128>>>(nStrips,noise_d,adc_d,nSeedStripsNC_d);
getNCSeedStrips<<<128,256>>>(nStrips,noise_d,adc_d,stripId_d,seedStripMask,seedStripsNCMask);
hipDeviceSynchronize();
getNCSeedStrips1<<<128,256>>>(nStrips,stripId_d,seedStripMask,seedStripsNCMask);
hipDeviceSynchronize();
//for( int i =0;i<nStrips;i++){
//printf("NCMask i n: %d %d\n",i, seedStripsNCMask[i]);
//}

nSeedStripsNC=0;
for(int l=0; l<nStrips;l++){
nSeedStripsNC += seedStripsNCMask[l];
//printf("mask[%d]: %d\n",l,seedStripMask[l]);
//printf("mask[%d]: %d\n",l,nSeedStripsNC_d[l]);
}
//printf("done %d\n",nSeedStripsNC);
//std::cout<<"nStrips "<<nStrips<<"nSeedStrips "<<nSeedStrips<<"nSeedStripsNC "<<nSeedStripsNC<<std::endl;

//printf("test 1:%d\n",nSeedStripsNC_d[439133]);
//cudaMemcpy(nSeedStripsNC_out, nSeedStripsNC_d,nStrips*sizeof(int),cudaMemcpyDeviceToHost);


//for (int j=0; j< nStrips; j++){ 
//nSeedStripsNC += nSeedStripsNCMask[j];
//}

//printf("test 2");
  int *seedStripsNCIndex ;//= (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
  int *clusterLastIndexLeft ;//= (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
  int *clusterLastIndexRight ;//= (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
  float *clusterNoiseSquared;// = (float *)_mm_malloc(nSeedStripsNC*sizeof(float), IDEAL_ALIGNMENT);
  uint8_t *clusterADCs ;//= (uint8_t *)_mm_malloc(nSeedStripsNC*256*sizeof(uint8_t), IDEAL_ALIGNMENT);
  bool *trueCluster;//= (bool *)_mm_malloc(nSeedStripsNC*sizeof(bool), IDEAL_ALIGNMENT);

  hipMallocManaged((void**)&seedStripsNCIndex,nSeedStripsNC*sizeof(int));
  hipMallocManaged((void**)&clusterLastIndexLeft,nSeedStripsNC*sizeof(int));
  hipMallocManaged((void**)&clusterLastIndexRight,nSeedStripsNC*sizeof(int));
  hipMallocManaged((void**)&clusterNoiseSquared,nSeedStripsNC*sizeof(float));
  hipMallocManaged((void**)&clusterADCs,nSeedStripsNC*256*sizeof(uint8_t));
  hipMallocManaged((void**)&trueCluster,nSeedStripsNC*sizeof(bool));
  int j=0;
  for (int i=0; i<nStrips; i++) {
    if (seedStripsNCMask[i] == true) {
      seedStripsNCIndex[j] = i;
      j++;
    }
  }

  if (j!=nSeedStripsNC) {
    std::cout<<"j "<<j<<"nSeedStripsNC "<<nSeedStripsNC<<std::endl;
    exit (1);
  }
//for(int l=0; l<10000;l++){
//printf("stripsNCMask[%d]: %d\n",l,seedStripsNCIndex[l]);
//}
//int k =0;
//for (int j=0; j< nStrips; j++){ 
//if( nSeedStripsNC_d[j] == 1.0){
//nSeedStripsNC += 1;
//seedStripsNCIndex[k] = j;
//k++;
//}
//}
//printf("test 3");
//printf("test x: %d\n",nSeedStripsNC);
//  int j=0;
//  for (int i=0; i<nStrips; i++) {
//    if (seedStripNCMask[i] == true) {
//      seedStripsNCIndex[j] = i;
//      j++;
//    }
//  }
//  if (j!=nSeedStripsNC) {
//    std::cout<<"j "<<j<<" nSeedStripsNC "<<nSeedStripsNC<<std::endl;
//    exit (1);
//  }

//  for (int i=0; i<nSeedStripsNC; i++) {
//    trueCluster[i] = false;
//    clusterNoiseSquared[i] = 0;
//  }

  // find the left and right bounday of the candidate cluster
  // (currently, we assume no bad strip. fix later)
//printf("test y\n");


findBoundries<<<128,256>>>(nStrips, nSeedStripsNC,seedStripsNCIndex,clusterNoiseSquared,stripId_d,clusterLastIndexLeft,clusterLastIndexRight,adc_d,noise_d);
hipDeviceSynchronize();
printf("complete\n");
//for(int l=0; l<10000;l++){
//printf("clusterLIL[%d]: %d\n",l,clusterLastIndexRight[l]);
//}

//printf("test z\n");
//  for (int i=0; i<nSeedStripsNC; i++) {
//    clusterNoiseSquared[i] = 0.0;
//    int index=seedStripsNCIndex[i];
//    clusterLastIndexLeft[i] = index;
//    clusterLastIndexRight[i] = index;
//    //uint8_t adc_i = adc[index];
//    float noise_i = noise[index];
//    clusterNoiseSquared[i] += noise_i*noise_i;
//    // find left boundary
//    int testIndex=index-1;
//    while(index>0&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndex]-1)>=0)&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndex]-1)<=MaxSequentialHoles)){
//   // while(index>0&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndex]-1)<=MaxSequentialHoles)){
//      float testNoise = noise[testIndex];
//      uint8_t testADC = adc[testIndex];
//      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
//	--clusterLastIndexLeft[i];
//	clusterNoiseSquared[i] += testNoise*testNoise;
//      }
//      --testIndex;
//    }
//
//    // find right boundary
//    testIndex=index+1;
//    while(testIndex<nStrips&&((stripId[testIndex]-stripId[clusterLastIndexRight[i]]-1)>=0)&&((stripId[testIndex]-stripId[clusterLastIndexRight[i]]-1)<=MaxSequentialHoles)){
////    while(testIndex<nStrips&&((stripId[testIndex]-stripId[clusterLastIndexRight[i]]-1)<=MaxSequentialHoles)) {
//      float testNoise = noise[testIndex];
//      uint8_t testADC = adc[testIndex];
//      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
//        ++clusterLastIndexRight[i];
//	clusterNoiseSquared[i] += testNoise*testNoise;
//      }
//      ++testIndex;
//    }
//  }


clusterChecker<<<128,256>>>(nSeedStripsNC,clusterLastIndexLeft,clusterLastIndexRight,adc_d, clusterNoiseSquared,gain_d, clusterADCs, trueCluster);
hipDeviceSynchronize();
printf("complete2\n");
//for(int l=0; l<1000;l++){
//printf("cluster[%d]: %d\n",l,trueCluster[l]);
//}
//  // check if the candidate cluster is a true cluster
//  // if so, do some adjustment for the adc values
//  for (int i=0; i<nSeedStripsNC; i++){
//    int left=clusterLastIndexLeft[i];
//    int right=clusterLastIndexRight[i];
//    int size=right-left+1;
//    int adcsum = 0;
//    for (int j=0; j<size; j++) {
//      adcsum += (int)adc[left+j];
//    }
//    bool noiseSquaredPass = clusterNoiseSquared[i]*ClusterThresholdSquared <= ((float)(adcsum)*float(adcsum));
//    bool chargePerCMPass = (float)(adcsum)/0.047f > minGoodCharge;
//    if (noiseSquaredPass&&chargePerCMPass) {
//      for (int j=0; j<size; j++){
//	uint8_t adc_j = adc[left+j];
//	float gain_j = gain[left+j];
//	auto charge = int( float(adc_j)/gain_j + 0.5f );
//	if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
//	clusterADCs[j*nSeedStripsNC+i] = adc_j;
//      }
//      trueCluster[i] = true;
//    }
//  }
//
  // print out the result
  gettimeofday(&timecheck, NULL);
  end = (double)timecheck.tv_sec *1000 + (double)timecheck.tv_usec/1000;
printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
  for (int i=0; i<nSeedStripsNC; i++) {
    
    if (trueCluster[i]){
    //printf("test 1\n");
      int index = clusterLastIndexLeft[i];
      std::cout<</*"cluster "<<i<<*/" det Id "<<detId[index]<<" fed Id "<<fedId[index]<<" strip "<<stripId[clusterLastIndexLeft[i]]<<" ADC ";
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      int size=right-left+1;
      for (int j=0; j<size; j++){
	std::cout<<(int)clusterADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }


  printf("time: %e (ms)\n",(end-start));
	
  free(detId);
  free(fedId);
  free(stripId);
  free(adc);
  free(noise);
  hipFree(adc_d);
  hipFree(noise_d);
  hipFree(gain_d);
  hipFree(stripId_d);
  free(gain);
  free(bad);
//  free(seedStripMask);
//  free(seedStripNCMask);
//  free(seedStripsNCIndex);
//  free(clusterNoiseSquared);
//  free(clusterLastIndexLeft);
//  free(clusterLastIndexRight);
//  free(clusterADCs);
//  free(trueCluster);

  return 0;

}
