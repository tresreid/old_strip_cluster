//  nvcc -arch=sm_35 -rdc=true cuda-strip-cluster.cu -o cuda_strip 


#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <mm_malloc.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#define IDEAL_ALIGNMENT 64
using detId_t = uint32_t;
using fedId_t = uint16_t;
using fedCh_t = uint8_t;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void getNCSeedStrips(const int nStrips,const float* noise_d,const uint16_t* adc,const int SeedThreshold,int* nSeedStripsNC_old)
{

  // find the seed strips
   unsigned int in = threadIdx.x + blockIdx.x*blockDim.x+1;
while(in<nStrips){
    float noise_i = noise_d[in];
    uint8_t adc_i = static_cast<uint8_t>(adc[in]);
    float noise_iR = noise_d[in-1];
    uint8_t adc_iR = static_cast<uint8_t>(adc[in-1]);
     nSeedStripsNC_old[in] = ((adc_iR >= static_cast<uint8_t>( noise_iR * SeedThreshold)) && (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)));
     printf("xxx: %d %d\n",in, ((adc_iR >= static_cast<uint8_t>( noise_iR * SeedThreshold)) && (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold))));
    //seedStripMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? true:false;
 // printf("test 5: %d\n",in);
    //if (seedStripMask[i] == true) nSeedStrips++;

in = in + blockDim.x*gridDim.x;
  }
//__syncthreads();
//if(in==0){// nSeedStripsNC_old = nSeedStripsNCx;
//printf("done");
//}
//  int nSeedStripsNC=0;
//  for (int i=0; i<nStrips; i++) {
//    if (seedStripMask[i] == true) {
//      if (stripId[i]-stripId[i-1]!=1) {
//	seedStripNCMask[i] = true;
//	nSeedStripsNC++;
  //    }
  //  }
 // }
//in += blockDim.x*gridDim.x;
}



int main()
{

  double start, end;
  struct timeval timecheck;
  gettimeofday(&timecheck, NULL);
  start = (double)timecheck.tv_sec *1000 + (double)timecheck.tv_usec /1000;

  int max_strips = 1400000;
  detId_t *detId = (detId_t *)_mm_malloc(max_strips*sizeof(detId_t), IDEAL_ALIGNMENT);
  fedId_t *fedId = (fedId_t *)_mm_malloc(max_strips*sizeof(fedId_t), IDEAL_ALIGNMENT);
  fedCh_t *fedCh = (fedCh_t *)_mm_malloc(max_strips*sizeof(fedCh_t), IDEAL_ALIGNMENT);
  uint16_t *stripId = (uint16_t *)_mm_malloc(max_strips*sizeof(uint16_t), IDEAL_ALIGNMENT);
  uint16_t *adc = (uint16_t *)_mm_malloc(max_strips*sizeof(uint16_t), IDEAL_ALIGNMENT);
  float *noise = (float *)_mm_malloc(max_strips*sizeof(float), IDEAL_ALIGNMENT);
  float *gain = (float *)_mm_malloc(max_strips*sizeof(float), IDEAL_ALIGNMENT);
  bool *bad = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);
  bool *seedStripMask = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);
  bool *seedStripNCMask = (bool *)_mm_malloc(max_strips*sizeof(bool), IDEAL_ALIGNMENT);

  //gpuErrchk((cudaMallocManaged((void**)adc,max_strips*sizeof(uint16_t))));
  //gpuErrchk((cudaMallocManaged((void**)noise,max_strips*sizeof(float))));

  // read in the data
  std::ifstream digidata_in("digidata.bin", std::ofstream::in | std::ios::binary);
  int i=0;
  while (digidata_in.read((char*)&detId[i], sizeof(detId_t)).gcount() == sizeof(detId_t)) {
    digidata_in.read((char*)&fedId[i], sizeof(fedId_t));
    digidata_in.read((char*)&fedCh[i], sizeof(fedCh_t));
    digidata_in.read((char*)&stripId[i], sizeof(uint16_t));
    digidata_in.read((char*)&adc[i], sizeof(uint16_t));
    digidata_in.read((char*)&noise[i], sizeof(float));
    digidata_in.read((char*)&gain[i], sizeof(float));
    digidata_in.read((char*)&bad[i], sizeof(bool));
    if (bad[i])
      std::cout<<"detid "<<detId[i]<<" fedId "<<fedId[i]<<" fedCh "<<(int)fedCh[i]<<" stripId "<<stripId[i]<<
      " adc "<<adc[i]<<" noise "<<noise[i]<<" gain "<<gain[i]<<" bad "<<bad[i]<<std::endl;

    i++;
  }
  int nStrips=i;
  

  float ChannelThreshold = 2.0, SeedThreshold = 3.0, ClusterThresholdSquared = 25.0;
  uint8_t MaxSequentialHoles = 0, MaxSequentialBad = 1, MaxAdjacentBad = 0;
  bool RemoveApvShots = true;
  float minGoodCharge = 1620.0;
  int nSeedStripsNC=0;
  int* nSeedStripsNC_d;
  int* nSeedStripsNC_out;
  //cudaMallocManaged((void*)nSeedStripsNC,sizeof(int));

//
//  // find the seed strips
//  int nSeedStrips=0;
//  for (int i=0; i<nStrips; i++) {
//    float noise_i = noise[i];
//    uint8_t adc_i = static_cast<uint8_t>(adc[i]);
//    seedStripMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? true:false;
//    if (seedStripMask[i] == true) nSeedStrips++;
//  }
//
//  int nSeedStripsNC=0;
//  for (int i=0; i<nStrips; i++) {
//    if (seedStripMask[i] == true) {
//      if (stripId[i]-stripId[i-1]!=1) {
//	seedStripNCMask[i] = true;
//	nSeedStripsNC++;
//      }
//    }
//  }
  float* noise_d;
  uint16_t* adc_d;
  hipMalloc((void**)&noise_d, sizeof(noise));
  hipMalloc((void**)&adc_d, sizeof(adc));
  //cudaMalloc((void**)&nSeedStripsNC_d, max_strips*sizeof(int));
  hipMallocManaged((void**)&nSeedStripsNC_d, nStrips*sizeof(int));
  //cudaalloc((void**)&nSeedStripsNC_out, max_strips*sizeof(int));
  //cudaMemcpy(&nSeedStripsNC_d, &nSeedStripsNC,sizeof(nSeedStripsNC),cudaMemcpyHostToDevice);
  hipMemcpy(noise_d, noise,sizeof(noise),hipMemcpyHostToDevice);
  hipMemcpy(adc_d, adc,sizeof(adc),hipMemcpyHostToDevice);
printf("test 0: %d\n", nStrips);
//for( int j =0; j<nStrips; j++){
//printf("noise[%d]: %f\n",j,noise[j]);} 
getNCSeedStrips<<<32,128>>>(nStrips,noise_d,adc_d,SeedThreshold,nSeedStripsNC_d);
hipDeviceSynchronize();
//std::cout<<"nStrips "<<nStrips<<"nSeedStrips "<<nSeedStrips<<"nSeedStripsNC "<<nSeedStripsNC<<std::endl;

printf("test 1:%d\n",nSeedStripsNC_d[439133]);
//cudaMemcpy(nSeedStripsNC_out, nSeedStripsNC_d,nStrips*sizeof(int),cudaMemcpyDeviceToHost);

printf("test 2");
for (int j=0; j< nStrips; j++){ nSeedStripsNC += nSeedStripsNC_d[j];}
//printf("test 3");
printf("test x: %d\n",nSeedStripsNC);
//  int *seedStripsNCIndex = (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
//  int *clusterLastIndexLeft = (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
//  int *clusterLastIndexRight = (int *)_mm_malloc(nSeedStripsNC*sizeof(int), IDEAL_ALIGNMENT);
//  float *clusterNoiseSquared = (float *)_mm_malloc(nSeedStripsNC*sizeof(float), IDEAL_ALIGNMENT);
//  uint8_t *clusterADCs = (uint8_t *)_mm_malloc(nSeedStripsNC*256*sizeof(uint8_t), IDEAL_ALIGNMENT);
//  bool *trueCluster= (bool *)_mm_malloc(nSeedStripsNC*sizeof(bool), IDEAL_ALIGNMENT);
//
//  int j=0;
//  for (int i=0; i<nStrips; i++) {
//    if (seedStripNCMask[i] == true) {
//      seedStripsNCIndex[j] = i;
//      j++;
//    }
//  }
//
//  if (j!=nSeedStripsNC) {
//    std::cout<<"j "<<j<<"nSeedStripsNC "<<nSeedStripsNC<<std::endl;
//    exit (1);
//  }
//
//  for (int i=0; i<nSeedStripsNC; i++) {
//    trueCluster[i] = false;
//    clusterNoiseSquared[i] = 0;
//  }
//
//  // find the left and right bounday of the candidate cluster
//  // (currently, we assume no bad strip. fix later)
//  for (int i=0; i<nSeedStripsNC; i++) {
//    int index=seedStripsNCIndex[i];
//    clusterLastIndexLeft[i] = index;
//    clusterLastIndexRight[i] = index;
//    uint8_t adc_i = adc[index];
//    float noise_i = noise[index];
//    clusterNoiseSquared[i] += noise_i*noise_i;
//    // find left boundary
//    int testIndex=index-1;
//    while(index>0&&((stripId[clusterLastIndexLeft[i]]-stripId[testIndex]-1)<=MaxSequentialHoles)){
//      float testNoise = noise[testIndex];
//      uint8_t testADC = adc[testIndex];
//      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
//	--clusterLastIndexLeft[i];
//	clusterNoiseSquared[i] += testNoise*testNoise;
//      }
//      --testIndex;
//    }
//
//    // find right boundary
//    testIndex=index+1;
//    while(testIndex<nStrips&&((stripId[testIndex]-stripId[clusterLastIndexRight[i]]-1)<=MaxSequentialHoles)) {
//      float testNoise = noise[testIndex];
//      uint8_t testADC = adc[testIndex];
//      if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
//        ++clusterLastIndexRight[i];
//	clusterNoiseSquared[i] += testNoise*testNoise;
//      }
//      ++testIndex;
//    }
//  }
//
//  // check if the candidate cluster is a true cluster
//  // if so, do some adjustment for the adc values
//  for (int i=0; i<nSeedStripsNC; i++){
//    int left=clusterLastIndexLeft[i];
//    int right=clusterLastIndexRight[i];
//    int size=right-left+1;
//    int adcsum = 0;
//    for (int j=0; j<size; j++) {
//      adcsum += (int)adc[left+j];
//    }
//    bool noiseSquaredPass = clusterNoiseSquared[i]*ClusterThresholdSquared <= ((float)(adcsum)*float(adcsum));
//    bool chargePerCMPass = (float)(adcsum)/0.047f > minGoodCharge;
//    if (noiseSquaredPass&&chargePerCMPass) {
//      for (int j=0; j<size; j++){
//	uint8_t adc_j = adc[left+j];
//	float gain_j = gain[left+j];
//	auto charge = int( float(adc_j)/gain_j + 0.5f );
//	if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
//	clusterADCs[j*nSeedStripsNC+i] = adc_j;
//      }
//      trueCluster[i] = true;
//    }
//  }
//
//  // print out the result
//  for (int i=0; i<nSeedStripsNC; i++) {
//    if (trueCluster[i]){
//      int index = clusterLastIndexLeft[i];
//      std::cout<<"cluster "<<i<<" det Id "<<detId[index]<<" fed Id "<<fedId[index]<<" strip "<<stripId[clusterLastIndexLeft[i]]<<" ADC ";
//      int left=clusterLastIndexLeft[i];
//      int right=clusterLastIndexRight[i];
//      int size=right-left+1;
//      for (int j=0; j<size; j++){
//	std::cout<<(int)clusterADCs[j*nSeedStripsNC+i]<<" ";
//      }
//      std::cout<<std::endl;
//    }
//  }
//

  gettimeofday(&timecheck, NULL);
  end = (double)timecheck.tv_sec *1000 + (double)timecheck.tv_usec/1000;
  printf("time: %e (ms)\n",(end-start));
	
  free(detId);
  free(fedId);
  free(stripId);
  free(adc);
  free(noise);
  hipFree(adc_d);
  hipFree(noise_d);
  free(gain);
  free(bad);
//  free(seedStripMask);
//  free(seedStripNCMask);
//  free(seedStripsNCIndex);
//  free(clusterNoiseSquared);
//  free(clusterLastIndexLeft);
//  free(clusterLastIndexRight);
//  free(clusterADCs);
// free(trueCluster);

  return 0;

}
